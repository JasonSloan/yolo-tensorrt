#include <cmath>
#include <hip/hip_runtime.h>

static __device__ void affine_project(float* matrix, float x, float y, float* ox, float* oy){
    *ox = matrix[0] * x + matrix[1] * y + matrix[2];
    *oy = matrix[3] * x + matrix[4] * y + matrix[5];
}

static __global__ void decode_kernel(
    float* predict, int num_bboxes, int num_classes, float confidence_threshold, 
    float* invert_affine_matrix, float* parray, int max_objects, int NUM_BOX_ELEMENT
){  
    int position = blockDim.x * blockIdx.x + threadIdx.x;
    if (position >= num_bboxes) return;

    float* pitem     = predict + (5 + num_classes) * position;
    float objectness = pitem[4];
    if(objectness < confidence_threshold)
        return;

    float* class_confidence = pitem + 5;
    float confidence        = *class_confidence++;
    int label               = 0;
    for(int i = 1; i < num_classes; ++i, ++class_confidence){
        if(*class_confidence > confidence){
            confidence = *class_confidence;
            label      = i;
        }
    }

    confidence *= objectness;
    if(confidence < confidence_threshold)
        return;

    int index = atomicAdd(parray, 1);
    if(index >= max_objects)
        return;

    float cx         = *pitem++;
    float cy         = *pitem++;
    float width      = *pitem++;
    float height     = *pitem++;
    float left   = cx - width * 0.5f;
    float top    = cy - height * 0.5f;
    float right  = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;
    // affine_project(invert_affine_matrix, left,  top,    &left,  &top);
    // affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

    // left, top, right, bottom, confidence, class, keepflag
    float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++ = left;
    *pout_item++ = top;
    *pout_item++ = right;
    *pout_item++ = bottom;
    *pout_item++ = confidence;
    *pout_item++ = label;
    *pout_item++ = 1; // 1 = keep, 0 = ignore
}

static __device__ float box_iou(
    float aleft, float atop, float aright, float abottom, 
    float bleft, float btop, float bright, float bbottom
){

    float cleft 	= max(aleft, bleft);
    float ctop 		= max(atop, btop);
    float cright 	= min(aright, bright);
    float cbottom 	= min(abottom, bbottom);
    
    float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
    if(c_area == 0.0f)
        return 0.0f;
    
    float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
    float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
    return c_area / (a_area + b_area - c_area);
}

static __global__ void fast_nms_kernel(float* bboxes, int max_objects, float threshold, int NUM_BOX_ELEMENT){

    int position = (blockDim.x * blockIdx.x + threadIdx.x);
    int count = min((int)*bboxes, max_objects);
    if (position >= count) 
        return;
    
    // left, top, right, bottom, confidence, class, keepflag
    float* pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
    for(int i = 0; i < count; ++i){
        float* pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
        if(i == position || pcurrent[5] != pitem[5]) continue;

        if(pitem[4] >= pcurrent[4]){
            if(pitem[4] == pcurrent[4] && i < position)
                continue;

            float iou = box_iou(
                pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                pitem[0],    pitem[1],    pitem[2],    pitem[3]
            );

            if(iou > threshold){
                pcurrent[6] = 0;  // 1=keep, 0=ignore
                return;
            }
        }
    }
} 

void postprocess_kernel_invoker(
    float* predict, int num_bboxes, int num_classes, float confidence_threshold, 
    float nms_threshold, float* invert_affine_matrix, float* parray, int max_objects, int NUM_BOX_ELEMENT, hipStream_t stream){
    
    int threads_per_block1 = 256;
    int blocks_per_grid1 = (num_bboxes + threads_per_block1 - 1) / threads_per_block1;
    dim3 cudaBlockSize1(threads_per_block1, 1, 1);
    dim3 cudaGridSize1(blocks_per_grid1, 1, 1);

    /* 如果核函数有波浪线，没关系，他是正常的，你只是看不顺眼罢了 */
    decode_kernel<<<cudaGridSize1, cudaBlockSize1, 0, stream>>>(
        predict, num_bboxes, num_classes, confidence_threshold, 
        invert_affine_matrix, parray, max_objects, NUM_BOX_ELEMENT
    );

    int threads_per_block2 = 256;
    int blocks_per_grid2 = (max_objects + threads_per_block2 - 1) / threads_per_block2;
    dim3 cudaBlockSize2(threads_per_block2, 1, 1);
    dim3 cudaGridSize2(blocks_per_grid2, 1, 1);
    
    fast_nms_kernel<<<cudaGridSize2, cudaBlockSize2, 0, stream>>>(parray, max_objects, nms_threshold, NUM_BOX_ELEMENT);
}